#include "hip/hip_runtime.h"
#include "oneflow/customized/kernels/scalar_binary_kernel.h"
#include "oneflow/core/device/cuda_util.h"

namespace oneflow {

namespace {

template<template<typename> class binary_func, typename T>
__global__ void ScalarBinaryRightGpu(const T* in_ptr, const T scalar_operand, T* out_ptr,
                                     const int64_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) { out_ptr[i] = binary_func<T>::Invoke(in_ptr[i], scalar_operand); }
}
template<template<typename> class binary_func, typename T>
__global__ void ScalarBinaryRightGpuInplace(T* in_ptr, const T scalar_operand, const int64_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) { in_ptr[i] = binary_func<T>::Invoke(in_ptr[i], scalar_operand); }
}
template<template<typename> class binary_func, typename T>
__global__ void ScalarBinaryLeftGpu(const T* in_ptr, const T scalar_operand, T* out_ptr,
                                    const int64_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) { out_ptr[i] = binary_func<T>::Invoke(scalar_operand, in_ptr[i]); }
}
template<template<typename> class binary_func, typename T>
__global__ void ScalarBinaryLeftGpuInplace(T* in_ptr, const T scalar_operand, const int64_t n) {
  CUDA_1D_KERNEL_LOOP(i, n) { in_ptr[i] = binary_func<T>::Invoke(scalar_operand, in_ptr[i]); }
}
}  // namespace

template<template<typename> class binary_func, typename T>
class LeftBinaryKernel<binary_func, DeviceType::kGPU, T> final : public user_op::OpKernel {
 public:
  LeftBinaryKernel() = default;
  ~LeftBinaryKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* in_ptr = GetInPtr<T>(ctx);
    auto* out_ptr = GetOutPtr<T>(ctx);
    const auto scalar_operand = GetScalarOperand<T>(ctx);
    const auto n = GetElemCnt(ctx);

    if (out_ptr == in_ptr) {
      RUN_CUDA_KERNEL((ScalarBinaryLeftGpuInplace<binary_func, T>), ctx->device_ctx(), n, out_ptr,
                      scalar_operand, n);
    } else {
      RUN_CUDA_KERNEL((ScalarBinaryLeftGpu<binary_func, T>), ctx->device_ctx(), n, in_ptr,
                      scalar_operand, out_ptr, n);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

template<template<typename> class binary_func, typename T>
class RightBinaryKernel<binary_func, DeviceType::kGPU, T> final : public user_op::OpKernel {
 public:
  RightBinaryKernel() = default;
  ~RightBinaryKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const auto* in_ptr = GetInPtr<T>(ctx);
    auto* out_ptr = GetOutPtr<T>(ctx);
    const auto scalar_operand = GetScalarOperand<T>(ctx);
    const auto n = GetElemCnt(ctx);

    if (out_ptr == in_ptr) {
      RUN_CUDA_KERNEL((ScalarBinaryRightGpuInplace<binary_func, T>), ctx->device_ctx(), n, out_ptr,
                      scalar_operand, n);
    } else {
      RUN_CUDA_KERNEL((ScalarBinaryRightGpu<binary_func, T>), ctx->device_ctx(), n, in_ptr,
                      scalar_operand, out_ptr, n);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_KERNEL(op_name, kernel_type, func_name, kernel_device_type, dtype)         \
  REGISTER_USER_KERNEL(op_name)                                                             \
      .SetCreateFn<                                                                         \
          kernel_type##BinaryKernel<func_name, DeviceType::k##kernel_device_type, dtype>>() \
      .SetIsMatchedPred([](const user_op::KernelRegContext& ctx) {                          \
        const user_op::TensorDesc* y_desc = ctx.TensorDesc4ArgNameAndIndex("out", 0);       \
        return ctx.device_type() == DeviceType::k##kernel_device_type                       \
               && y_desc->data_type() == GetDataType<dtype>::value;                         \
      });

#define REGISTER_ADD_KERNEL_WITH_TYPE(type, _) \
  REGISTER_KERNEL("scalar_add", Commutative, BinaryFuncAdd, GPU, type)

OF_PP_FOR_EACH_TUPLE(REGISTER_ADD_KERNEL_WITH_TYPE, ARITHMETIC_DATA_TYPE_SEQ)

#undef REGISTER_ADD_KERNEL_WITH_TYPE

#define ARITHMETIC_DATA_TYPE_SEQ_WITHOUT_INT8     \
  OF_PP_MAKE_TUPLE_SEQ(int32_t, DataType::kInt32) \
  OF_PP_MAKE_TUPLE_SEQ(int64_t, DataType::kInt64) \
  FLOATING_DATA_TYPE_SEQ

#define REGISTER_MUL_DIV_KERNEL_WITH_TYPE(type, _)                     \
  REGISTER_KERNEL("scalar_mul", Commutative, BinaryFuncMul, GPU, type) \
  REGISTER_KERNEL("left_scalar_div", Left, BinaryFuncDiv, GPU, type)   \
  REGISTER_KERNEL("right_scalar_div", Right, BinaryFuncDiv, GPU, type)

// OF_PP_FOR_EACH_TUPLE(REGISTER_MUL_DIV_KERNEL_WITH_TYPE, ARITHMETIC_DATA_TYPE_SEQ_WITHOUT_INT8)

#undef REGISTER_MUL_DIV_KERNEL_WITH_TYPE
#undef ARITHMETIC_DATA_TYPE_SEQ_WITHOUT_INT8

}  // namespace oneflow
