#include "oneflow/core/operator/deconv_op.h"
#include "oneflow/core/kernel/deconv_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/kernel/kernel.h"
#include "oneflow/core/device/cudnn_util.h"
#include "oneflow/core/device/cudnn_conv_util.h"

namespace oneflow {

template<typename T>
class DeconvGPUKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(DeconvGPUKernel);
  DeconvGPUKernel() = default;
  ~DeconvGPUKernel() = default;

 private:
  const PbMessage& GetCustomizedOpConf() const override { return this->op_conf().deconv_conf(); }

  void VirtualKernelInit() override {
    const DeconvOpConf& op_conf = this->op_conf().deconv_conf();
    const ConvConf& conv_conf = this->op_conf().deconv_conf().conv_conf();
    const int32_t num_spatial_dims = this->op_conf().deconv_conf().conv_conf().num_spatial_dims();
    DenseShapeView x_shape(this->kernel_conf().deconv_conf().in());
    DenseShapeView y_shape(this->kernel_conf().deconv_conf().out());
    DenseShapeView weight_shape(this->kernel_conf().deconv_conf().weight());

    const std::string& data_format = conv_conf.data_format();
    this->x_desc_.reset(new CudnnTensorDesc(GetDataType<T>::value, x_shape, data_format));
    this->y_desc_.reset(new CudnnTensorDesc(GetDataType<T>::value, y_shape, data_format));
    this->filter_desc_.reset(new CudnnFilterDesc(GetDataType<T>::value, weight_shape, data_format));
    this->deconv_desc_.reset(new CudnnDeconvDesc(GetDataType<T>::value, x_shape,
                                                 this->op_conf().deconv_conf()));
  }

  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    CudnnConvArgs args(this->op_conf().deconv_conf().conv_conf(), ctx.device_ctx->cudnn_handle(),
                       BnInOp2Blob("y"), BnInOp2Blob("x"), BnInOp2Blob("filter"),
                       BnInOp2Blob("cudnn_buf"),
                       this->job_desc().job_conf().cudnn_conv_use_deterministic_algo_only(),
                       this->job_desc().job_conf().cudnn_conv_heuristic_search_algo());
    hipdnnConvolutionBwdDataAlgo_t algo;
    size_t work_space_size = 0;
    if (this->job_desc().job_conf().has_cudnn_conv_force_bwd_data_algo()) {
      algo = static_cast<hipdnnConvolutionBwdDataAlgo_t>(
          this->job_desc().job_conf().cudnn_conv_force_bwd_data_algo());
      CudaCheck(GetConvWorkspaceSize(args, algo, &work_space_size));
    } else {
      auto algo_perf = FindCudnnConvAlgorithm<hipdnnConvolutionBwdDataAlgoPerf_t>(args);
      algo = algo_perf->algo;
      work_space_size = algo_perf->memory;
    }
    CHECK_LE(work_space_size, BnInOp2Blob("cudnn_buf")->ByteSizeOfBlobBody());
    CudaCheck(hipdnnConvolutionBackwardData(args.handle, CudnnSPOnePtr<T>(), args.wdesc.Get(),
                                           args.w_dptr, args.ydesc.Get(), args.y_dptr,
                                           args.cdesc.Get(), algo, args.work_space, work_space_size,
                                           CudnnSPZeroPtr<T>(), args.xdesc.Get(), args.x_dptr));
  }

  mutable std::unique_ptr<CudnnTensorDesc> x_desc_;
  mutable std::unique_ptr<CudnnTensorDesc> y_desc_;
  mutable std::unique_ptr<CudnnFilterDesc> filter_desc_;
  mutable std::unique_ptr<CudnnDeconvDesc> deconv_desc_;
};

#define REGISTER_DECONV_GPU_KERNEL(dtype)                                                   \
  REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kDeconvConf, DeviceType::kGPU, dtype, \
                                        DeconvGPUKernel<dtype>)

REGISTER_DECONV_GPU_KERNEL(float);
REGISTER_DECONV_GPU_KERNEL(double);
REGISTER_DECONV_GPU_KERNEL(float16);

}  //  namespace oneflow
