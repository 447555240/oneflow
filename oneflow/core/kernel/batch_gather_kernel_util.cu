#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/batch_gather_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include <assert.h>

namespace oneflow {

namespace {

template<typename K, typename IDX>
__device__ int64_t GetInOffset(const IDX out_offset, const K* indices, const IDX indices_num,
                               const IDX instance_size, const IDX gather_dim_size) {
  const IDX batch_idx = out_offset / (indices_num * instance_size);
  const IDX indices_idx = out_offset % (indices_num * instance_size) / instance_size;
  const IDX inner_idx = out_offset % instance_size;
  const K idx = indices[batch_idx * indices_num + indices_idx];
  assert(idx >= 0 && idx < gather_dim_size);
  return batch_idx * gather_dim_size * instance_size + idx * instance_size + inner_idx;
}

template<typename T, typename K>
__global__ void BatchGatherForwardGpu(const int64_t elem_cnt, const T* in, const K* indices,
                                      const int64_t indices_num, const int64_t instance_size,
                                      const int64_t gather_dim_size, T* out) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    out[i] = in[GetInOffset<K, int32_t>(i, indices, indices_num, instance_size, gather_dim_size)];
  }
}

}  // namespace

template<typename T, typename K>
struct BatchGatherKernelUtilImpl<DeviceType::kGPU, T, K> final {
  static void Forward(DeviceCtx* ctx, const T* in, const K* indices, const Shape& flat_out_shape,
                      const int64_t gather_dim_size, T* out);
};

template<typename T, typename K>
void BatchGatherKernelUtilImpl<DeviceType::kGPU, T, K>::Forward(DeviceCtx* ctx, const T* in,
                                                                const K* indices,
                                                                const Shape& flat_out_shape,
                                                                const int64_t gather_dim_size,
                                                                T* out) {
  const int64_t batch_num = flat_out_shape.At(0);
  const int64_t indices_num = flat_out_shape.At(1);
  const int64_t instance_size = flat_out_shape.At(2);

  const int64_t elem_cnt = batch_num * indices_num * instance_size;
  BatchGatherForwardGpu<T, K>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          elem_cnt, in, indices, indices_num, instance_size, gather_dim_size, out);
}

#define INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU(in_type_pair, index_type_pair)          \
  template struct BatchGatherKernelUtilImpl<DeviceType::kGPU, OF_PP_PAIR_FIRST(in_type_pair), \
                                            OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU,
                                 FLOATING_DATA_TYPE_SEQ, INT_DATA_TYPE_SEQ);
#undef INSTANTIATE_BATCH_GATHER_KERNEL_UTIL_IMPL_GPU

}  // namespace oneflow
