#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/cuda_copy_peer_kernel_util.h"
#include "oneflow/core/kernel/kernel_util.cuh"

#define N_THREAD 1024
#define N_LOOP 16

namespace oneflow {

namespace {

__forceinline__ __device__ int32_t DivUp(int32_t n, int32_t val) { return (n + val - 1) / val; }

__forceinline__ __device__ void Fetch(ulong2& v, const ulong2* p) {
  // clang-format off
  asm volatile("ld.volatile.global.v2.u64 {%0,%1}, [%2];" : "=l"(v.x), "=l"(v.y) : "l"(p) : "memory");
  // clang-format on
}

__forceinline__ __device__ void Store(ulong2* p, ulong2& v) {
  // clang-format off
  asm volatile("st.volatile.global.v2.u64 [%0], {%1,%2};" :: "l"(p), "l"(v.x), "l"(v.y) : "memory");
  // clang-format on
}

__global__ void ReadKernel(void* buf, const void* src, volatile int32_t* step_mutex, size_t size) {
  const int32_t step_size = N_THREAD * N_LOOP * sizeof(ulong2);
  const int32_t n_step = DivUp(size, step_size);
  const int32_t thread_id = threadIdx.x;
  if (thread_id == 0) { assert(*step_mutex == 0); }
  __syncthreads();
  for (int32_t step = 0; step < n_step; ++step) {
    int32_t step_offset = step * step_size;
    ulong2 v;
#pragma unroll
    for (int32_t l = 0; l < N_LOOP; ++l) {
      const int32_t offset = step_offset + (l * N_THREAD + thread_id) * sizeof(ulong2);
      if (offset < size) {
        Fetch(v, reinterpret_cast<const ulong2*>(static_cast<const uint8_t*>(src) + offset));
        Store(reinterpret_cast<ulong2*>(static_cast<uint8_t*>(buf) + offset), v);
      }
    }
    __syncthreads();
    __threadfence_system();
    if (thread_id == 0) { *step_mutex = step + 1; }
  }
}

__global__ void WriteKernel(void* dst, const void* buf, volatile int32_t* step_mutex, size_t size) {
  const int32_t step_size = N_THREAD * N_LOOP * sizeof(ulong2);
  const int32_t n_step = DivUp(size, step_size);
  const int32_t thread_id = threadIdx.x;
  __syncthreads();
  for (int32_t step = 0; step < n_step; ++step) {
    if (thread_id == 0) {
      const int32_t next_step = step + 1;
      while (*step_mutex < next_step) {}
    }
    __syncthreads();
    __threadfence_system();
    int32_t step_offset = step * step_size;
    ulong2 v;
#pragma unroll
    for (int32_t l = 0; l < N_LOOP; ++l) {
      const int32_t offset = step_offset + (l * N_THREAD + thread_id) * sizeof(ulong2);
      if (offset < size) {
        Fetch(v, reinterpret_cast<const ulong2*>(static_cast<const uint8_t*>(buf) + offset));
        Store(reinterpret_cast<ulong2*>(static_cast<uint8_t*>(dst) + offset), v);
      }
    }
  }
}

}  // namespace

void CudaCopyPeerKernelUtil::CopyAsync(void* dst, void* buf, const void* src, int32_t* step_mutex,
                                       size_t size, int32_t dst_dev_id, int32_t src_dev_id,
                                       hipStream_t read, hipStream_t write) {
  dim3 dim_grid(1, 1, 1);
  dim3 dim_block(N_THREAD, 1, 1);
  struct hipLaunchParams params[2];
  void* read_kernel_args[] = {(void*)(&buf), (void*)(&src), (void*)(&step_mutex), (void*)(&size)};
  void* write_kernel_args[] = {(void*)(&dst), (void*)(&buf), (void*)(&step_mutex), (void*)(&size)};
  params[0].func = (void*)ReadKernel;
  params[0].gridDim = dim_grid;
  params[0].blockDim = dim_block;
  params[0].sharedMem = 0;
  params[0].args = read_kernel_args;
  params[0].stream = read;

  params[1].func = (void*)WriteKernel;
  params[1].gridDim = dim_grid;
  params[1].blockDim = dim_block;
  params[1].sharedMem = 0;
  params[1].args = write_kernel_args;
  params[1].stream = write;

  CudaCheck(hipLaunchCooperativeKernelMultiDevice(
      params, 2,
      hipCooperativeLaunchMultiDeviceNoPreSync | hipCooperativeLaunchMultiDeviceNoPostSync));
}

}  // namespace oneflow
