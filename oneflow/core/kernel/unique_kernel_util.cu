#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/unique_kernel_util.h"
#include <hipcub/hipcub.hpp>

namespace oneflow {

namespace {

template<typename T>
struct Buffer final {
  T* ptr = nullptr;
  size_t size_in_bytes = 0;
};

int64_t SizeAlign(int64_t size) { return RoundUp(size, kCudaAlignSize); }

template<typename T, typename U>
int64_t GetSortKeySize(int64_t n) {
  return SizeAlign(n * sizeof(T));
}

template<typename T, typename U>
int64_t GetSortValueSize(int64_t n) {
  return SizeAlign(n * sizeof(U));
}

template<typename T, typename U>
int64_t GetCubSortTempStorageSize(int64_t n) {
  size_t cub_sort_temp_store_size = 0;
  CudaCheck(hipcub::DeviceRadixSort::SortPairs<T, U>(nullptr, cub_sort_temp_store_size, nullptr,
                                                  nullptr, nullptr, nullptr, n));
  CHECK_GE(cub_sort_temp_store_size, 0);
  CHECK_LT(cub_sort_temp_store_size, GetMaxVal<int64_t>());
  return SizeAlign(static_cast<int64_t>(cub_sort_temp_store_size));
}

template<typename T, typename U>
int64_t GetCubRleTempStorageSize(int64_t n) {
  size_t cub_rle_temp_store_size = 0;
  CudaCheck(hipcub::DeviceRunLengthEncode::Encode<T*, T*, U*, int64_t*>(
      nullptr, cub_rle_temp_store_size, nullptr, nullptr, nullptr, nullptr, n));
  CHECK_GE(cub_rle_temp_store_size, 0);
  CHECK_LT(cub_rle_temp_store_size, GetMaxVal<int64_t>());
  return SizeAlign(static_cast<int64_t>(cub_rle_temp_store_size));
}

template<typename T, typename U>
int64_t GetCubTempStorageSize(int64_t n) {
  const int64_t sort_temp_storage_size = GetCubSortTempStorageSize<T, U>(n);
  const int64_t rle_temp_storage_size = GetCubRleTempStorageSize<T, U>(n);
  return std::max(sort_temp_storage_size, rle_temp_storage_size);
}

template<typename T>
void AliasPtr(void* origin, int64_t* offset, Buffer<T>* buffer, int64_t size) {
  auto* ptr = reinterpret_cast<unsigned char*>(origin);
  if (buffer != nullptr) {
    buffer->ptr = reinterpret_cast<T*>(ptr + *offset);
    buffer->size_in_bytes = size;
  }
  *offset += size;
}

template<typename T, typename U>
void UniqueAliasWorkspace(DeviceCtx* ctx, int64_t n, void* workspace,
                          int64_t* workspace_size_in_bytes, Buffer<T>* cub_sort_keys_out,
                          Buffer<U>* cub_sort_values_in, Buffer<U>* cub_sort_values_out,
                          Buffer<void>* cub_temp_storage) {
  int64_t offset = 0;
  AliasPtr(workspace, &offset, cub_sort_keys_out, GetSortKeySize<T, U>(n));
  AliasPtr(workspace, &offset, cub_sort_values_in, GetSortValueSize<T, U>(n));
  AliasPtr(workspace, &offset, cub_sort_values_out, GetSortValueSize<T, U>(n));
  AliasPtr(workspace, &offset, cub_temp_storage, GetCubTempStorageSize<T, U>(n));
  *workspace_size_in_bytes = offset;
}

template<typename T>
__global__ void IotaKernel(int64_t n, T* out) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = static_cast<T>(i); }
}

}  // namespace

template<typename T, typename U>
struct UniqueKernelUtil<DeviceType::kGPU, T, U> {
  static void Unique(DeviceCtx* ctx, int64_t n, const T* in, int64_t* num_unique, T* unique_out,
                     U* idx_out, void* workspace, int64_t workspace_size_in_bytes);
  static void GetUniqueWorkspaceSizeInBytes(DeviceCtx* ctx, int64_t n,
                                            int64_t* workspace_size_in_bytes);
};

template<typename T, typename U>
void UniqueKernelUtil<DeviceType::kGPU, T, U>::Unique(DeviceCtx* ctx, int64_t n, const T* in,
                                                      int64_t* num_unique, T* unique_out,
                                                      U* idx_out, void* workspace,
                                                      int64_t workspace_size_in_bytes) {
  int64_t rt_workspace_size;
  Buffer<T> cub_sort_keys_out;
  Buffer<U> cub_sort_values_in;
  Buffer<U> cub_sort_values_out;
  Buffer<void> cub_temp_storage;
  UniqueAliasWorkspace<T, U>(ctx, n, workspace, &rt_workspace_size, &cub_sort_keys_out,
                             &cub_sort_values_in, &cub_sort_values_out, &cub_temp_storage);
  IotaKernel<U><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
      n, cub_sort_values_in.ptr);
  CudaCheck(hipcub::DeviceRadixSort::SortPairs<T, U>(
      cub_temp_storage.ptr, cub_temp_storage.size_in_bytes, in, cub_sort_keys_out.ptr,
      cub_sort_values_in.ptr, cub_sort_values_out.ptr, n, 0, sizeof(T) * 8, ctx->hip_stream()));
}

template<typename T, typename U>
void UniqueKernelUtil<DeviceType::kGPU, T, U>::GetUniqueWorkspaceSizeInBytes(
    DeviceCtx* ctx, int64_t n, int64_t* workspace_size_in_bytes) {
  UniqueAliasWorkspace<T, U>(ctx, n, nullptr, workspace_size_in_bytes, nullptr, nullptr, nullptr,
                             nullptr);
}

#define INSTANTIATE_UNIQUE_KERNEL_UTIL_GPU(k_type_pair, v_type_pair)                \
  template struct UniqueKernelUtil<DeviceType::kGPU, OF_PP_PAIR_FIRST(k_type_pair), \
                                   OF_PP_PAIR_FIRST(v_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_UNIQUE_KERNEL_UTIL_GPU, UNIQUE_KERNEL_KV_DATA_TYPE_SEQ,
                                 UNIQUE_KERNEL_KV_DATA_TYPE_SEQ);
#undef INSTANTIATE_UNIQUE_KERNEL_UTIL_GPU

}  // namespace oneflow
