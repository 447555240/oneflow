#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/indexed_slices_lazy_adam_optimizer_kernel_util.h"
#include "oneflow/core/kernel/unique_kernel_util.h"
#include "oneflow/core/kernel/normal_model_update_kernel.cuh"

namespace oneflow {

namespace {

template<typename T, typename K>
__global__ void UpdateModelGpu(T l1, T l2, T beta1, T beta2, T epsilon, int64_t feature_size,
                               const int64_t* num_unique_instance, const int64_t* train_step,
                               const float* learning_rate, const K* indices, const T* values,
                               T* model, T* m, T* v) {
  const T beta1_t = pow(beta1, *train_step);
  const T beta2_t = pow(beta2, *train_step);
  const float local_learning_rate = *learning_rate * sqrt(1 - (beta2_t)) / (1 - (beta1_t));
  const int64_t n = *num_unique_instance * feature_size;
  CUDA_1D_KERNEL_LOOP(i, n) {
    const K instance_id = indices[i / feature_size];
    const T diff = values[i];
    const K model_idx = instance_id * feature_size + i % feature_size;
    const T old_model = model[model_idx];
    T reg_diff = RegDiff(diff, l1, l2, old_model);
    m[model_idx] = beta1 * m[model_idx] + (1 - beta1) * reg_diff;
    v[model_idx] = beta2 * v[model_idx] + (1 - beta2) * reg_diff * reg_diff;
    model[model_idx] =
        old_model - local_learning_rate * m[model_idx] / (sqrt(v[model_idx]) + epsilon);
  }
}

}  // namespace

template<typename T, typename K>
struct IndexedSlicesLazyAdamOptimizerKernelUtil<DeviceType::kGPU, T, K> {
  static void UpdateModel(DeviceCtx* ctx, T l1, T l2, T beta1, T beta2, T epsilon,
                          int64_t num_instance, int64_t feature_size,
                          const int64_t* num_unique_instance, const int64_t* train_step,
                          const float* learning_rate, const K* indices, const T* values, T* model,
                          T* m, T* v);
};

template<typename T, typename K>
void IndexedSlicesLazyAdamOptimizerKernelUtil<DeviceType::kGPU, T, K>::UpdateModel(
    DeviceCtx* ctx, T l1, T l2, T beta1, T beta2, T epsilon, int64_t num_instance,
    int64_t feature_size, const int64_t* num_unique_instance, const int64_t* train_step,
    const float* learning_rate, const K* indices, const T* values, T* model, T* m, T* v) {
  UpdateModelGpu<T, K><<<BlocksNum4ThreadsNum(num_instance * feature_size), kCudaThreadsNumPerBlock,
                         0, ctx->hip_stream()>>>(l1, l2, beta1, beta2, epsilon, feature_size,
                                                  num_unique_instance, train_step, learning_rate,
                                                  indices, values, model, m, v);
}

#define INSTANTIATE_INDEXED_SLICES_LAZY_ADAM_OPTIMIZER_KERNEL_UTIL_GPU(key_type_pair, \
                                                                       idx_type_pair) \
  template struct IndexedSlicesLazyAdamOptimizerKernelUtil<                           \
      DeviceType::kGPU, OF_PP_PAIR_FIRST(key_type_pair), OF_PP_PAIR_FIRST(idx_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_INDEXED_SLICES_LAZY_ADAM_OPTIMIZER_KERNEL_UTIL_GPU,
                                 UNIQUE_KERNEL_KV_DATA_TYPE_SEQ, UNIQUE_KERNEL_KV_DATA_TYPE_SEQ);
#undef INSTANTIATE_INDEXED_SLICES_LAZY_ADAM_OPTIMIZER_KERNEL_UTIL_GPU

}  // namespace oneflow
