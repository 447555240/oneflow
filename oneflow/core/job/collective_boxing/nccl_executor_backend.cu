#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/job/collective_boxing/nccl_executor_backend.h"
#include "oneflow/core/job/collective_boxing/request_store.h"
#include "oneflow/core/device/nccl_util.h"
#include "oneflow/core/graph/boxing/collective_boxing_util.h"
#include "oneflow/core/job/resource_desc.h"
#include "oneflow/core/job/machine_context.h"
#include "oneflow/core/control/ctrl_client.h"
#include "oneflow/core/kernel/batch_memcpy_kernel_util.h"
#include "oneflow/core/job/global_for.h"
#include "oneflow/core/thread/thread_pool.h"
#include "oneflow/core/device/cuda_util.h"
#include <nccl.h>

#include <memory>

namespace oneflow {

namespace boxing {

namespace collective {

namespace {

ncclRedOp_t GetNcclReduceOp(ReduceMethod reduce_method) {
  if (reduce_method == kReduceMethodSum) {
    return ncclRedOp_t::ncclSum;
  } else {
    UNIMPLEMENTED();
  }
}

std::string GetNcclUniqueIdRpcKey(const std::string& name, int64_t stream_id) {
  return "CollectiveBoxingExecutorNcclUniqueIdRpcKey-" + name + "-" + std::to_string(stream_id);
}

int64_t GetRequestSize(const RequestDesc& request) {
  return Shape(request.op_desc().shape()).elem_cnt()
         * GetSizeOfDataType(request.op_desc().data_type());
}

int64_t GetAlignedRequestSize(const RequestDesc& request) {
  return GetCudaAlignedSize(GetRequestSize(request));
}

struct CopyParams {
  void* dst;
  const void* src;
  int64_t count;
};

constexpr int64_t kMultiCopyParamsMaxSize = 128;

struct MultiCopyParams {
  CopyParams params[kMultiCopyParamsMaxSize];
  int64_t count;

  MultiCopyParams() : count(0), params{} {}

  void Add(void* dst, const void* src, int64_t count) {
    CHECK_LT(this->count, kMultiCopyParamsMaxSize);
    params[this->count].dst = dst;
    params[this->count].src = src;
    params[this->count].count = count;
    this->count += 1;
  }
};

using BulkType = ulonglong2;

__global__ void MultiCopyGpu(MultiCopyParams multi_params) {
  for (int64_t p = 0; p < multi_params.count; ++p) {
    const CopyParams params = multi_params.params[p];
    auto* bulk_dst = reinterpret_cast<BulkType*>(params.dst);
    const auto* bulk_src = reinterpret_cast<const BulkType*>(params.src);
    const int64_t bulk_count = params.count / sizeof(BulkType);
    CUDA_1D_KERNEL_LOOP_T(int64_t, i, bulk_count) { bulk_dst[i] = bulk_src[i]; }
    const int64_t tail_offset = bulk_count * sizeof(BulkType);
    auto* tail_dst = reinterpret_cast<char*>(params.dst) + tail_offset;
    const auto* tail_src = reinterpret_cast<const char*>(params.src) + tail_offset;
    const int64_t tail_count = params.count - tail_offset;
    CUDA_1D_KERNEL_LOOP_T(int64_t, i, tail_count) { tail_dst[i] = tail_src[i]; }
  }
}

void MultiCopy(hipStream_t stream, const MultiCopyParams& multi_params) {
  if (multi_params.count <= 0) { return; }
  CHECK_LE(multi_params.count, kMultiCopyParamsMaxSize);
  int64_t max_count = multi_params.params[0].count;
  for (int64_t i = i; i < multi_params.count; ++i) {
    max_count = std::max(max_count, multi_params.params[i].count);
  }
  MultiCopyGpu<<<BlocksNum4ThreadsNum(max_count), kCudaThreadsNumPerBlock, 0, stream)>>>(
      multi_params);
}

class CommRank final {
 public:
  OF_DISALLOW_COPY(CommRank);
  CommRank(int32_t device_id, int32_t global_rank, int32_t global_rank_count, int32_t local_rank,
           int32_t local_rank_count)
      : device_id_(device_id),
        global_rank_(global_rank),
        global_rank_count_(global_rank_count),
        local_rank_(local_rank),
        local_rank_count_(local_rank_count),
        nccl_comm_(nullptr) {}

  ~CommRank() {
    if (nccl_comm_ != nullptr) {
      CudaCurrentDeviceGuard(device_id_);
      OF_NCCL_CHECK(ncclCommDestroy(nccl_comm_));
    }
  }

  int32_t device_id() const { return device_id_; }

  int32_t global_rank() const { return global_rank_; }

  int32_t global_rank_count() const { return global_rank_count_; }

  int32_t local_rank() const { return local_rank_; }

  int32_t local_rank_count() const { return local_rank_count_; }

  void InitRank(ncclUniqueId unique_id) {
    CudaCurrentDeviceGuard(device_id_);
    OF_NCCL_CHECK(ncclCommInitRank(&nccl_comm_, global_rank_count_, unique_id, global_rank_));
  }

 private:
  int32_t device_id_;
  int32_t global_rank_;
  int32_t global_rank_count_;
  int32_t local_rank_;
  int32_t local_rank_count_;
  ncclComm_t nccl_comm_;
};

class CommGroup final {
 public:
  OF_DISALLOW_COPY(CommGroup);
  CommGroup() = default;
  ~CommGroup() = default;

  void InitGroup(const DeviceSet& device_set, const std::string& unique_name) {
    const int64_t this_machine_id = Global<MachineCtx>::Get()->this_machine_id();
    global_rank_count_ = device_set.device_size();
    std::vector<int32_t> local_ranks;
    for (int32_t i = 0; i < global_rank_count_; ++i) {
      if (device_set.device(i).machine_id() == this_machine_id) { local_ranks.push_back(i); }
    }
    const int32_t local_rank_count = local_ranks.size();
    CHECK_GT(local_rank_count, 0);
    ncclUniqueId nccl_unique_id{};
    if (local_ranks.front() == 0) {
      if (local_rank_count != global_rank_count_) {
        Global<CtrlClient>::Get()->PushKV(unique_name, NcclUniqueIdToString(nccl_unique_id));
      }
      OF_NCCL_CHECK(ncclGetUniqueId(&nccl_unique_id));
    } else {
      Global<CtrlClient>::Get()->PullKV(unique_name, [&nccl_unique_id](const std::string& val) {
        NcclUniqueIdFromString(val, &nccl_unique_id);
      });
    }
    rank_vec_.reserve(local_rank_count);
    OF_NCCL_CHECK(ncclGroupStart());
    for (int32_t local_rank = 0; local_rank < local_ranks.size(); ++local_rank) {
      const int32_t global_rank = local_ranks.at(local_rank);
      const int32_t device_id = device_set.device(global_rank).device_id();
      OF_CUDA_CHECK(hipSetDevice(device_id));
      rank_vec_.emplace_back(device_id, global_rank, global_rank_count_, local_rank,
                             local_rank_count);
      rank_vec_.at(local_rank).InitRank(nccl_unique_id);
    }
    OF_NCCL_CHECK(ncclGroupEnd());
  }

  int32_t global_rank_count() const { return global_rank_count_; }

  int32_t local_rank_count() const { return rank_vec_.size(); }

 private:
  std::vector<CommRank> rank_vec_;
  int32_t global_rank_count_ = 0;
};

class StreamCtx {
 public:
  OF_DISALLOW_COPY(StreamCtx);
};

};  // namespace

struct NcclExecutorBackend::Impl {
  HashMap<DeviceSet, std::vector<CommGroup>> device_set2stream_id2comm_group;
};

NcclExecutorBackend::NcclExecutorBackend()
    : collective_boxing_conf_(Global<ResourceDesc, ForSession>::Get()->collective_boxing_conf()),
      shutdown_(false) {
  OF_CUDA_CHECK(hipGetDeviceCount(&num_devices_));
  callback_executor_pool_.reset(new ThreadPool(num_devices_));
  CHECK_GT(collective_boxing_conf_.nccl_num_streams(), 0);
  num_streams_ = collective_boxing_conf_.nccl_num_streams();
  CHECK_GE(collective_boxing_conf_.nccl_fusion_threshold_mb(), 0);
  fusion_threshold_ = collective_boxing_conf_.nccl_fusion_threshold_mb() * 1024 * 1024;
  event_list_poll_thread_ = std::thread([this]() {
    std::list<Event> local_event_list;
    while (true) {
      {
        std::unique_lock<std::mutex> lock(event_list_mutex_);
        if (local_event_list.empty()) {
          event_list_cond_.wait(lock, [this]() { return (!event_list_.empty()) || shutdown_; });
        }
        local_event_list.splice(local_event_list.end(), event_list_);
      }
      if (local_event_list.empty() && shutdown_) { break; }
      for (auto it = local_event_list.begin(); it != local_event_list.end();) {
        OF_CUDA_CHECK(hipSetDevice(it->device_id));
        hipError_t err = hipEventQuery(it->cuda_event);
        if (err == hipErrorNotReady) {
          ++it;
          continue;
        } else if (err == hipSuccess) {
          OF_CUDA_CHECK(hipEventDestroy(it->cuda_event));
          auto callback_ptr =
              std::make_shared<std::function<void(Maybe<void>)>>(std::move(it->callback));
          callback_executor_pool_->AddWork(
              [callback_ptr]() { (*callback_ptr)(Maybe<void>::Ok()); });
          local_event_list.erase(it++);
        } else {
          OF_CUDA_CHECK(err);
          UNIMPLEMENTED();
        }
      }
    }
  });
}

NcclExecutorBackend::~NcclExecutorBackend() {
  {
    std::unique_lock<std::mutex> lock(event_list_mutex_);
    shutdown_ = true;
    event_list_cond_.notify_all();
  }
  event_list_poll_thread_.join();
  callback_executor_pool_.reset();
  CudaCurrentDeviceGuard guard;
  for (auto& device_id2device_ctx : stream_id2device_id2device_ctx_) {
    for (auto& device_id7device_ctx : device_id2device_ctx) {
      OF_CUDA_CHECK(hipSetDevice(device_id7device_ctx.first));
      OF_CUDA_CHECK(hipStreamSynchronize(device_id7device_ctx.second->stream));
      OF_CUDA_CHECK(hipStreamDestroy(device_id7device_ctx.second->stream));
      OF_CUDA_CHECK(hipFree(device_id7device_ctx.second->fusion_buffer));
    }
  }
  for (auto& device_set7stream_id2device_id2comm : device_set2stream_id2device_id2comm_) {
    for (auto& device_id2comm : device_set7stream_id2device_id2comm.second) {
      for (auto& device_id7comm : device_id2comm) {
        OF_CUDA_CHECK(hipSetDevice(device_id7comm.first));
        OF_NCCL_CHECK(ncclCommDestroy(device_id7comm.second));
      }
    }
  }
}

void NcclExecutorBackend::GroupRequests(const std::vector<int32_t>& request_ids,
                                        std::vector<std::vector<int32_t>>* groups) {
  std::vector<int32_t> group;
  int64_t group_size = 0;
  auto IsOpFusionEnabled = [&](const RequestDesc& request) -> bool {
    const OpType op_type = request.op_desc().op_type();
    if (op_type == OpType::kOpTypeAllReduce) {
      return collective_boxing_conf_.nccl_fusion_all_reduce();
    } else if (op_type == OpType::kOpTypeAllGather) {
      return collective_boxing_conf_.nccl_fusion_all_gather();
    } else if (op_type == OpType::kOpTypeReduceScatter) {
      return collective_boxing_conf_.nccl_fusion_reduce_scatter();
    } else if (op_type == OpType::kOpTypeReduce) {
      return collective_boxing_conf_.nccl_fusion_reduce();
    } else if (op_type == OpType::kOpTypeBroadcast) {
      return collective_boxing_conf_.nccl_fusion_broadcast();
    } else if (op_type == OpType::kOpTypeAll2All) {
      return false;
    } else {
      UNIMPLEMENTED();
      return false;
    }
  };
  auto CanFuse = [&](const RequestDesc& lhs, const RequestDesc& rhs) -> bool {
    const bool enable_mixed_fusion = (!collective_boxing_conf_.nccl_fusion_all_reduce_use_buffer())
                                     && collective_boxing_conf_.nccl_enable_mixed_fusion();
    if (lhs.device_set() != rhs.device_set()) { return false; }
    if (!IsOpFusionEnabled(lhs) || !IsOpFusionEnabled(rhs)) { return false; }
    if (lhs.op_desc().op_type() != rhs.op_desc().op_type() && (!enable_mixed_fusion)) {
      return false;
    }
    const OpType op_type = lhs.op_desc().op_type();
    if (op_type == OpType::kOpTypeAllReduce) {
      if (collective_boxing_conf_.nccl_fusion_all_reduce_use_buffer()) {
        CHECK(lhs.op_desc().has_reduce_method());
        CHECK(rhs.op_desc().has_reduce_method());
        return lhs.op_desc().reduce_method() == rhs.op_desc().reduce_method()
               && lhs.op_desc().data_type() == rhs.op_desc().data_type();
      } else {
        return true;
      }
    } else if (op_type == OpType::kOpTypeReduce || op_type == OpType::kOpTypeBroadcast
               || op_type == OpType::kOpTypeReduceScatter || op_type == OpType::kOpTypeAllGather) {
      return true;
    } else if (op_type == OpType::kOpTypeAll2All) {
      return false;
    } else {
      UNIMPLEMENTED();
      return false;
    }
  };

  for (const int32_t request_id : request_ids) {
    const auto& request = request_store_->MutRequestEntry(request_id)->desc();
    const int64_t size = GetAlignedRequestSize(request);
    if (group.empty() || !CanFuse(request_store_->MutRequestEntry(group.back())->desc(), request)
        || group_size + size > fusion_threshold_
        || group.size() >= collective_boxing_conf_.nccl_fusion_max_ops()) {
      if (!group.empty()) {
        groups->emplace_back();
        groups->back().swap(group);
        group_size = 0;
      }
    }
    group.push_back(request_id);
    group_size += size;
  }
  if (!group.empty()) {
    groups->emplace_back();
    groups->back().swap(group);
  }
}

void NcclExecutorBackend::ExecuteRequests(const std::vector<int32_t>& request_ids) {
  std::vector<const RequestDesc*> group;
  std::vector<std::map<int64_t, std::shared_ptr<const RuntimeRequestInfo>>> ranks;
  group.reserve(request_ids.size());
  for (const int32_t request_id : request_ids) {
    auto* request_entry = request_store_->MutRequestEntry(request_id);
    group.push_back(&request_entry->desc());
    ranks.emplace_back();
    for (int32_t local_rank = 0; local_rank < request_entry->LocalRankCount(); ++local_rank) {
      ranks.back()[request_entry->LocalRankToGlobalRank(local_rank)] =
          request_entry->GetRuntimeRequest(local_rank);
    }
    request_entry->ResetRuntimeRequest();
  }
  CHECK_EQ(group.size(), ranks.size());
  if (group.empty()) { return; }
  const int64_t group_size = group.size();
  std::map<int64_t, std::vector<std::shared_ptr<const std::function<void(const Maybe<void>&)>>>>
      device_id2callbacks;
  const int64_t stream_id = current_stream_id_;
  current_stream_id_ = (current_stream_id_ + 1) % num_streams_;
  CudaCurrentDeviceGuard device_guard;
  auto& device_id2comm =
      device_set2stream_id2device_id2comm_.size() == 1
          ? device_set2stream_id2device_id2comm_.begin()->second.at(stream_id)
          : device_set2stream_id2device_id2comm_.at(group.front()->device_set()).at(stream_id);
  auto& device_id2device_ctx = stream_id2device_id2device_ctx_.at(stream_id);
  if (group.front()->op_desc().op_type() == OpType::kOpTypeAllReduce
      && collective_boxing_conf_.nccl_fusion_all_reduce_use_buffer() && group.size() > 1) {
    int64_t offset = 0;
    std::map<int64_t, std::vector<MemcpyParam>> device_id2copy_in_params;
    std::map<int64_t, std::vector<MemcpyParam>> device_id2copy_out_params;
    for (int64_t i = 0; i < group.size(); ++i) {
      const RequestDesc* request_desc = group.at(i);
      if (i != 0) {
        CHECK_EQ(request_desc->op_desc().reduce_method(), group.front()->op_desc().reduce_method());
        CHECK_EQ(request_desc->op_desc().data_type(), group.front()->op_desc().data_type());
      }
      const std::map<int64_t, std::shared_ptr<const RuntimeRequestInfo>>& rank2request_info =
          ranks.at(i);
      const int64_t size = GetRequestSize(*request_desc);
      CHECK_LE(offset + size, fusion_threshold_);
      const int64_t aligned_size = GetCudaAlignedSize(size);
      for (const auto& rank7request_info : rank2request_info) {
        const int64_t rank = rank7request_info.first;
        const RuntimeRequestInfo& request_info = *rank7request_info.second;
        const DeviceDesc& device_desc = request_desc->device_set().device().Get(rank);
        const int64_t device_id = device_desc.device_id();
        auto& device_ctx = device_id2device_ctx.at(device_id);
        device_id2copy_in_params[device_id].push_back(MemcpyParam{
            .dst = device_ctx->fusion_buffer + offset,
            .src = request_info.send_buff,
            .count = static_cast<size_t>(size),
        });
        device_id2copy_out_params[device_id].push_back(MemcpyParam{
            .dst = request_info.recv_buff,
            .src = device_ctx->fusion_buffer + offset,
            .count = static_cast<size_t>(size),
        });
        device_id2callbacks[device_id].reserve(group_size);
        device_id2callbacks[device_id].push_back(request_info.callback);
      }
      offset += aligned_size;
    }
    for (auto& device_id7copy_in_params : device_id2copy_in_params) {
      OF_CUDA_CHECK(hipSetDevice(device_id7copy_in_params.first));

      BatchMemcpyKernelUtil<DeviceType::kGPU>::Copy(
          device_id2device_ctx.at(device_id7copy_in_params.first).get(),
          device_id7copy_in_params.second);
    }
    OF_NCCL_CHECK(ncclGroupStart());
    const int64_t size_of_data_type = GetSizeOfDataType(group.front()->op_desc().data_type());
    CHECK_EQ(offset % size_of_data_type, 0);
    const int64_t elem_cnt = offset / size_of_data_type;
    for (auto& device_id7comm : device_id2comm) {
      OF_CUDA_CHECK(hipSetDevice(device_id7comm.first));
      auto& device_ctx = device_id2device_ctx.at(device_id7comm.first);
      OF_NCCL_CHECK(ncclAllReduce(device_ctx->fusion_buffer, device_ctx->fusion_buffer, elem_cnt,
                                  GetNcclDataType(group.front()->op_desc().data_type()),
                                  GetNcclReduceOp(group.front()->op_desc().reduce_method()),
                                  device_id7comm.second, device_ctx->stream));
    }
    OF_NCCL_CHECK(ncclGroupEnd());
    for (auto& device_id7copy_out_params : device_id2copy_out_params) {
      OF_CUDA_CHECK(hipSetDevice(device_id7copy_out_params.first));
      BatchMemcpyKernelUtil<DeviceType::kGPU>::Copy(
          device_id2device_ctx.at(device_id7copy_out_params.first).get(),
          device_id7copy_out_params.second);
    }
  } else {
    OF_NCCL_CHECK(ncclGroupStart());
    for (int64_t i = 0; i < group.size(); ++i) {
      const RequestDesc* request_desc = group.at(i);
      const OpDesc& op_desc = request_desc->op_desc();
      const std::map<int64_t, std::shared_ptr<const RuntimeRequestInfo>>& rank2request_info =
          ranks.at(i);
      for (const auto& rank7request_info : rank2request_info) {
        const int64_t rank = rank7request_info.first;
        const RuntimeRequestInfo& request_info = *rank7request_info.second;
        const DeviceDesc& device_desc = request_desc->device_set().device().Get(rank);
        const int64_t device_id = device_desc.device_id();
        OF_CUDA_CHECK(hipSetDevice(device_id));
        ncclComm_t comm = device_id2comm.at(device_id);
        auto& device_ctx = device_id2device_ctx.at(device_id);
        ncclDataType_t nccl_data_type = GetNcclDataType(op_desc.data_type());
        const OpType op_type = op_desc.op_type();
        const int64_t num_ranks = op_desc.num_ranks();
        const int64_t elem_cnt = Shape(op_desc.shape()).elem_cnt();
        const void* send_buff = request_info.send_buff;
        void* recv_buff = request_info.recv_buff;
        device_id2callbacks[device_id].reserve(group_size);
        device_id2callbacks[device_id].push_back(request_info.callback);
        if (op_type == OpType::kOpTypeAllReduce) {
          OF_NCCL_CHECK(ncclAllReduce(send_buff, recv_buff, elem_cnt, nccl_data_type,
                                      GetNcclReduceOp(op_desc.reduce_method()), comm,
                                      device_ctx->stream));
        } else if (op_type == OpType::kOpTypeAllGather) {
          CHECK_EQ(elem_cnt % num_ranks, 0);
          OF_NCCL_CHECK(ncclAllGather(send_buff, recv_buff, elem_cnt / num_ranks, nccl_data_type,
                                      comm, device_ctx->stream));
        } else if (op_type == OpType::kOpTypeReduceScatter) {
          CHECK_EQ(elem_cnt % num_ranks, 0);
          OF_NCCL_CHECK(ncclReduceScatter(send_buff, recv_buff, elem_cnt / num_ranks,
                                          nccl_data_type, GetNcclReduceOp(op_desc.reduce_method()),
                                          comm, device_ctx->stream));
        } else if (op_type == OpType::kOpTypeReduce) {
          OF_NCCL_CHECK(ncclReduce(send_buff, recv_buff, elem_cnt, nccl_data_type,
                                   GetNcclReduceOp(op_desc.reduce_method()), op_desc.root(), comm,
                                   device_ctx->stream));
        } else if (op_type == OpType::kOpTypeBroadcast) {
          OF_NCCL_CHECK(ncclBroadcast(send_buff, recv_buff, elem_cnt, nccl_data_type,
                                      op_desc.root(), comm, device_ctx->stream));
        } else if (op_type == OpType::kOpTypeAll2All) {
#if NCCL_VERSION_CODE > 2700
          const int64_t elem_per_rank = elem_cnt / num_ranks;
          const int64_t elem_per_chunk = elem_per_rank / num_ranks;
          const int64_t dtype_size = GetSizeOfDataType(op_desc.data_type());
          const int64_t chunk_size = elem_per_chunk * dtype_size;
          for (int64_t j = 0; j < num_ranks; ++j) {
            OF_NCCL_CHECK(ncclSend(reinterpret_cast<const void*>(
                                       reinterpret_cast<const char*>(send_buff) + j * chunk_size),
                                   elem_per_chunk, nccl_data_type, j, comm, device_ctx->stream));
            OF_NCCL_CHECK(ncclRecv(
                reinterpret_cast<void*>(reinterpret_cast<char*>(recv_buff) + j * chunk_size),
                elem_per_chunk, nccl_data_type, j, comm, device_ctx->stream));
          }
#else
          UNIMPLEMENTED();
#endif
        } else {
          UNIMPLEMENTED();
        }
      }
    }
    OF_NCCL_CHECK(ncclGroupEnd());
  }
  for (auto& device_id7callbacks : device_id2callbacks) {
    const int64_t device_id = device_id7callbacks.first;
    OF_CUDA_CHECK(hipSetDevice(device_id));
    hipEvent_t event;
    OF_CUDA_CHECK(hipEventCreateWithFlags(&event, hipEventDisableTiming));
    OF_CUDA_CHECK(hipEventRecord(event, device_id2device_ctx.at(device_id)->stream));
    {
      std::unique_lock<std::mutex> event_list_lock(event_list_mutex_);
      event_list_.emplace_back(Event{device_id, event, [=](const Maybe<void>& status) {
                                       for (const auto& callback : device_id7callbacks.second) {
                                         (*callback)(status);
                                       }
                                     }});
      event_list_cond_.notify_all();
    }
  }
}

void NcclExecutorBackend::Init(const CollectiveBoxingPlan& collective_boxing_plan,
                               std::shared_ptr<RequestStore> request_store) {
  request_store_ = request_store;
  impl_ = std::make_unique<Impl>();
  CudaCurrentDeviceGuard guard;
  std::set<int64_t> local_device_ids;
  for (int32_t request_id = 0; request_id < request_store_->RequestCount(); ++request_id) {
    auto* request_entry = request_store_->MutRequestEntry(request_id);
    const auto& request = request_entry->desc();
    if (request.op_desc().backend() != Backend::kBackendNCCL) { continue; }
    if (!request_entry->HasRankOnThisNode()) { continue; }
    const DeviceSet& device_set = request.device_set();
    if (device_set2stream_id2device_id2comm_.count(device_set) > 0) { continue; }
    auto& stream_id2device_id2comm = device_set2stream_id2device_id2comm_[device_set];
    stream_id2device_id2comm.resize(num_streams_);
    for (int32_t stream_id = 0; stream_id < num_streams_; ++stream_id) {
      auto& device_id2comm = stream_id2device_id2comm.at(stream_id);
      for (int32_t local_rank = 0; local_rank < request_entry->LocalRankCount(); ++local_rank) {
        const int64_t device_id = request_entry->LocalDeviceDesc(local_rank).device_id();
        device_id2comm.emplace(device_id, ncclComm_t{});
        local_device_ids.emplace(device_id);
      }
      ncclUniqueId nccl_unique_id{};
      if (request_entry->IsRootOnThisNode()) {
        OF_NCCL_CHECK(ncclGetUniqueId(&nccl_unique_id));
        if (request_entry->NodeCount() > 1) {
          const std::string rpc_key = GetNcclUniqueIdRpcKey(request.op_desc().name(), stream_id);
          Global<CtrlClient>::Get()->PushKV(rpc_key, NcclUniqueIdToString(nccl_unique_id));
        }
      } else {
        const std::string rpc_key = GetNcclUniqueIdRpcKey(request.op_desc().name(), stream_id);
        Global<CtrlClient>::Get()->PullKV(rpc_key, [&nccl_unique_id](const std::string& val) {
          NcclUniqueIdFromString(val, &nccl_unique_id);
        });
      }
      OF_NCCL_CHECK(ncclGroupStart());
      for (int32_t local_rank = 0; local_rank < request_entry->LocalRankCount(); ++local_rank) {
        const int64_t device_id = request_entry->LocalDeviceDesc(local_rank).device_id();
        OF_CUDA_CHECK(hipSetDevice(device_id));
        const int32_t global_rank = request_entry->LocalRankToGlobalRank(local_rank);
        OF_NCCL_CHECK(ncclCommInitRank(&device_id2comm.at(device_id), device_set.device_size(),
                                       nccl_unique_id, global_rank));
      }
      OF_NCCL_CHECK(ncclGroupEnd())
          << "To see more detail, please run OneFlow with system variable NCCL_DEBUG=INFO";
    }
  }
  int cuda_stream_greatest_priority;
  OF_CUDA_CHECK(hipDeviceGetStreamPriorityRange(nullptr, &cuda_stream_greatest_priority));
  stream_id2device_id2device_ctx_.resize(num_streams_);
  for (int64_t stream_id = 0; stream_id < num_streams_; ++stream_id) {
    auto& device_id2device_ctx_ = stream_id2device_id2device_ctx_.at(stream_id);
    for (const int64_t device_id : local_device_ids) {
      device_id2device_ctx_.emplace(device_id, std::make_unique<NcclDeviceCtx>());
    }
    for (const int64_t device_id : local_device_ids) {
      auto& device_ctx = device_id2device_ctx_.at(device_id);
      OF_CUDA_CHECK(hipSetDevice(device_id));
      OF_CUDA_CHECK(hipStreamCreateWithPriority(&device_ctx->stream, hipStreamNonBlocking,
                                                 cuda_stream_greatest_priority));
      OF_CUDA_CHECK(hipMalloc(&device_ctx->fusion_buffer, fusion_threshold_));
    }
  }
}

}  // namespace collective

}  // namespace boxing

}  // namespace oneflow
