#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"

namespace oneflow {

namespace {

template<typename T, typename K, int32_t N>
struct Param {
  const T* in;
  const K* in_size;
  T* out[N];
  K* out_size[N];
  int64_t range_start;
  int64_t num_out;
};

template<typename T, typename K>
__global__ void GetPartionBoundIndex(const int64_t n, const int64_t parallel_num,
                                     const int64_t num_classes_per_rank, const T* in_ptr,
                                     const K* in_size_ptr, K* out_ptr) {
  const K num = in_size_ptr[0];
  CUDA_1D_KERNEL_LOOP(i, num) {
    if (i != 0) {
      const T cur_in = in_ptr[i] / num_classes_per_rank;
      const T pre_in = in_ptr[i - 1] / num_classes_per_rank;
      if (cur_in > pre_in) {
#pragma unroll
        for (int32_t j = pre_in + 1; j <= cur_in; ++j) { out_ptr[j] = static_cast<K>(i); }
      }
    }
  }
  CUDA_1D_KERNEL_LOOP(i, parallel_num + 1) {
    const K first_in = in_ptr[0] / num_classes_per_rank;
    const K last_in = in_ptr[num - 1] / num_classes_per_rank;
    if (i <= first_in) {
      out_ptr[i] = 0;
    } else if (i > last_in) {
      out_ptr[i] = num;
    }
  }
}

template<typename T, typename K, int32_t N>
__global__ void PartitionGpu(const int64_t n, const int64_t parallel_num,
                             const int64_t num_classes_per_rank, const K* partion_bound_index,
                             Param<T, K, N> param) {
  const K num = param.in_size[0];
  CUDA_1D_KERNEL_LOOP(i, num) {
#pragma unroll
    for (int32_t j = 0; j < param.num_out; ++j) {
      const int32_t partion_bound_index_start = partion_bound_index[j];
      if (i >= partion_bound_index_start && i < partion_bound_index[j + 1]) {
        const int32_t lower_bound = (param.range_start + j) * num_classes_per_rank;
        param.out[j][i - partion_bound_index_start] = param.in[i] - lower_bound;
        break;
      }
    }
  }
  CUDA_1D_KERNEL_LOOP(i, param.num_out) {
    param.out_size[i][0] = partion_bound_index[i + 1] - partion_bound_index[i];
  }
}

}  // namespace

template<typename T, typename K>
class PartitionKernel final : public user_op::OpKernel {
 public:
  PartitionKernel() = default;
  ~PartitionKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* in = ctx->Tensor4ArgNameAndIndex("in", 0);
    const user_op::Tensor* in_size = ctx->Tensor4ArgNameAndIndex("in_size", 0);
    const int64_t elem_cnt = in->shape().elem_cnt();
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const int64_t parallel_num = ctx->Attr<int64_t>("parallel_num");
    const int64_t num_classes = ctx->Attr<int64_t>("num_classes");
    CHECK_EQ(num_classes % parallel_num, 0);
    const int64_t num_classes_per_rank = num_classes / parallel_num;
    CHECK_EQ(ctx->user_op_conf().output_size("out"), parallel_num);
    CHECK_EQ(ctx->user_op_conf().output_size("out_size"), parallel_num);
    GetPartionBoundIndex<T, K><<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                                 ctx->device_ctx()->hip_stream()>>>(
        elem_cnt, parallel_num, num_classes_per_rank, in->dptr<T>(), in_size->dptr<K>(),
        tmp_buffer->mut_dptr<K>());
    Param<T, K, 128> para;
    para.in = in->dptr<T>();
    para.in_size = in_size->dptr<K>();
    int64_t remain_size = parallel_num;
    int64_t output_id = 0;
    while (remain_size > 0) {
      para.range_start = output_id;
      int64_t num_out = 0;
      if (remain_size > 128) {
        remain_size -= 128;
        para.num_out = 128;
      } else {
        para.num_out = remain_size;
        remain_size = 0;
      }
      for (int32_t i = 0; i < para.num_out; ++i) {
        user_op::Tensor* out = ctx->Tensor4ArgNameAndIndex("out", output_id);
        user_op::Tensor* out_size = ctx->Tensor4ArgNameAndIndex("out_size", output_id);
        output_id++;
        para.out[i] = out->mut_dptr<T>();
        para.out_size[i] = out_size->mut_dptr<K>();
      }
      PartitionGpu<T, K, 128>
          <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
             ctx->device_ctx()->hip_stream()>>>(elem_cnt, parallel_num, num_classes_per_rank,
                                                 tmp_buffer->dptr<K>() + para.range_start, para);
    }
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_PARTITION_KERNEL(dtype, ktype)                                              \
  REGISTER_USER_KERNEL("partition")                                                          \
      .SetCreateFn<PartitionKernel<dtype, ktype>>()                                          \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                    \
                       & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value)       \
                       & (user_op::HobDataType("out_size", 0) == GetDataType<ktype>::value)) \
      .SetInferTmpSizeFn([](user_op::InferContext* ctx) {                                    \
        const int64_t parallel_num = ctx->Attr<int64_t>("parallel_num");                     \
        return GetCudaAlignedSize((parallel_num + 1) * sizeof(ktype));                       \
      });

REGISTER_PARTITION_KERNEL(int32_t, int32_t)
REGISTER_PARTITION_KERNEL(int64_t, int32_t)
REGISTER_PARTITION_KERNEL(int32_t, int64_t)
REGISTER_PARTITION_KERNEL(int64_t, int64_t)

}  // namespace oneflow
