
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/user/kernels/hardtanh_kernel.h"
namespace oneflow {

namespace {

template<template<typename> class Opt, typename T>
struct ElemwiseHardtanhFunctor<DeviceType::kGPU, Opt, T> final {
  void operator()(DeviceCtx* ctx, const int64_t elem_cnt, T min_val, T max_val, T* out,
                  const T* in) {
    OF_CUDA_CHECK(oneflow::cuda::elementwise::Unary(HardtanhFunctor<T>(min_val, max_val), elem_cnt,
                                                    out, in, ctx->cuda_stream()));
  }
};

template<template<typename> class Opt, typename T>
struct ElemwiseHardtanhGradFunctor<DeviceType::kGPU, Opt, T> final {
  void operator()(DeviceCtx* ctx, const int64_t elem_cnt, T min_val, T max_val, T* dx, const T* y,
                  const T* dy) {
    OF_CUDA_CHECK(oneflow::cuda::elementwise::Binary(HardtanhGradFunctor<T>(min_val, max_val),
                                                     elem_cnt, dx, y, dy, ctx->cuda_stream()));
  };
};

}  // namespace

REGISTER_HARDTANH_KERNELS(DeviceType::kGPU, half);
REGISTER_HARDTANH_KERNELS(DeviceType::kGPU, float);
REGISTER_HARDTANH_KERNELS(DeviceType::kGPU, double);

}  // namespace oneflow
#endif  // WITH_CUDA
