
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA
#include "oneflow/user/kernels/masked_fork_kernel.h"

namespace oneflow {

template<typename T>
__global__ void MaskedForkGpuKernel(const int64_t elem_cnt, const T* in, const int8_t* mask,
                                    T* out_true, T* out_false) {
  ForkLoopFunctor<T>()(elem_cnt, in, mask, out_true, out_false);
}

template<typename T>
struct MaskedForkFunctor<DeviceType::kGPU, T> final {
  void operator()(DeviceCtx* ctx, const int64_t elem_cnt, const T* in, const int8_t* mask,
                  T* out_true, T* out_false) {
    MaskedForkGpuKernel<T>
        <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
            elem_cnt, in, mask, out_true, out_false);
  }
};

REGISTER_MASKED_FORK_KERNEL(DeviceType::kGPU, int8_t);
REGISTER_MASKED_FORK_KERNEL(DeviceType::kGPU, int32_t);
REGISTER_MASKED_FORK_KERNEL(DeviceType::kGPU, int64_t);
REGISTER_MASKED_FORK_KERNEL(DeviceType::kGPU, float);
REGISTER_MASKED_FORK_KERNEL(DeviceType::kGPU, double);
}  // namespace oneflow
#endif  // WITH_CUDA
